#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
	Dslash base code using MPI + CUDA

	Copyright 2015 Koichi Shirahata

	Written by
		Koichi Shirahata

--------------------------------------------------------------------*/
/*--------------------------------------------------------------------
	Dslash base code using MPI

	Copyright 2009-2013 IBM Research - Tokyo, IBM Corporation

	Written by
		Jun Doi  (doichan@jp.ibm.com)

--------------------------------------------------------------------*/

#include "dslash_base.h"
#include <stdio.h>
#include <stdlib.h>
#include <complex.h>
#include <hip/hip_complex.h>

#include <sys/time.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <mpi.h>

#include <omp.h>

// #include "qcd.h"
// #include "qcd_mult.h"

    

QCDReal qcdtKappa[4];
static int qcdBaseIeo = 0;

__device__ QCDReal dqcdtKappa[4];

int qcdNx;
int qcdNy;
int qcdNz;
int qcdNt;
int qcdNxy;
int qcdNxyz;
int qcdNsite;

int qcdMyRank;
int qcdNProcs;

int qcdNetSize[4];
int qcdNetPos[4];
int qcdRankNeighbors[8];
int qcdSx;
int qcdSy;
int qcdSz;
int qcdSt;

__device__ int dqcdNx;
__device__ int dqcdNy;
__device__ int dqcdNz;
__device__ int dqcdNt;
__device__ int dqcdNxy;
__device__ int dqcdNxyz;
__device__ int dqcdNsite;

static QCDComplex* qcdSendBuf[8];
static QCDComplex* qcdRecvBuf[8];

static QCDComplex* pqcdSendBuf[8];
static QCDComplex* pqcdRecvBuf[8];

static QCDComplex* dqcdSendBuf[8];
static QCDComplex* dqcdRecvBuf[8];

static hipStream_t stream[8];
// static hipStream_t stream[9];

int qcdNumThreadsDivY[QCD_NUM_MAX_THREADS][2];
int qcdNumThreadsDivZ[QCD_NUM_MAX_THREADS][2];



typedef struct __qcd_RNG__
{
	int s;
	int e;
}QCDRng;

QCDRng qcdRngX[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

QCDRng qcdRngYOut[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
QCDRng qcdRngYIn[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
QCDRng qcdRngYInBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

QCDRng qcdRngZOut[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
QCDRng qcdRngZIn[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
QCDRng qcdRngZInBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

QCDRng qcdRngT[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
QCDRng qcdRngTBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

__device__ QCDRng dqcdRngX[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

__device__ QCDRng dqcdRngYOut[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
__device__ QCDRng dqcdRngYIn[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
__device__ QCDRng dqcdRngYInBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

__device__ QCDRng dqcdRngZOut[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
__device__ QCDRng dqcdRngZIn[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
__device__ QCDRng dqcdRngZInBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];

__device__ QCDRng dqcdRngT[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];
__device__ QCDRng dqcdRngTBnd[QCD_NUM_MAX_THREADS][QCD_NUM_MAX_THREADS];


__global__ void cuQCDDopr_MakeXPB(QCDComplex* pXP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pWP += (dqcdRngX[nid][tid].s + gtid) * dqcdNx;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,pWP,dqcdNsite,DOPRSET_SPIN);

		//for Plus boundary (send to minus)
		QCD_UXP_HALF(hv,v);

		QCDDopr_Store_Stride(hv,(pXP+i),(dqcdNy*dqcdNz*dqcdNt),DOPRSET_HSPIN);

		pWP += stride * dqcdNx;
	}
}

__global__ void cuQCDDopr_MakeXMB(QCDComplex* pXM,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex t[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pWM += (dqcdRngX[nid][tid].s + gtid) * dqcdNx;
	pUM += (dqcdRngX[nid][tid].s + gtid) * dqcdNx;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,pWM,dqcdNsite,DOPRSET_SPIN);
		QCDDopr_LoadGauge_Stride(u,pUM);

		//for Minus boundary (send to plus)
		QCD_UXM_HALF(t,v);
		QCD_MUL_UM(hv,u,t);

		QCDDopr_Store_Stride(hv,(pXM+i),(dqcdNy*dqcdNz*dqcdNt),DOPRSET_HSPIN);

		pWM += stride * dqcdNx;
		pUM += stride * dqcdNx;
	}
}

__global__ void cuQCDDopr_XPin(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pWP += dqcdRngX[nid][tid].s + gtid;
	pUP += dqcdRngX[nid][tid].s + gtid;
	pV += dqcdRngX[nid][tid].s + gtid;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e * dqcdNx; i += stride){
	        if(i % dqcdNx < dqcdNx - 1) {
		        QCDDopr_Load_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

			//uxp
			QCDDopr_Load_Stride(w,pWP,dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,pUP);

			QCD_UXP_HALF(tv,w);
			QCD_MUL_UP(hv,u,tv);
			QCD_UXP_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		}
		pV += stride;
		pUP += stride;
		pWP += stride;
	}
}

__global__ void cuQCDDopr_XMin(QCDComplex* pV,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	// int i,x;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pWM += dqcdRngX[nid][tid].s + gtid;
	pUM += dqcdRngX[nid][tid].s + gtid;
	pV += dqcdRngX[nid][tid].s + gtid;

	pV += 1;
	pUM += 1;
	pWM += 1;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e * dqcdNx; i += stride){
	        if(i % dqcdNx < dqcdNx - 1) {
		        QCDDopr_Load_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

			//uxm
			QCDDopr_Load_Stride(w,pWM,dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,pUM);
			QCD_UXM_HALF(tv,w);
			QCD_MUL_UM(hv,u,tv);
			QCD_UXM_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		}
		pV += stride;
		pUM += stride;
		pWM += stride;
	}
}

__global__ void cuQCDDopr_SetXPBnd(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pXP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pUP += (dqcdRngX[nid][tid].s + gtid) * dqcdNx + dqcdNx - 1;
	pV += (dqcdRngX[nid][tid].s + gtid) * dqcdNx + dqcdNx - 1;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		//uxp
		QCDDopr_Load_Stride(hv,(pXP + i),(dqcdNy*dqcdNz*dqcdNt),DOPRSET_HSPIN);
		QCDDopr_LoadGauge_Stride(u,pUP);

		QCD_MUL_UP(tv,u,hv);
		QCD_UXP_SET(v,tv,kappa);

		QCDDopr_Store_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		pV += stride * dqcdNx;
		pUP += stride * dqcdNx;
	}
}

__global__ void cuQCDDopr_SetXMBnd(QCDComplex* pV,QCDComplex* pXM,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV += (dqcdRngX[nid][tid].s + gtid) * dqcdNx;
	for(i = gtid + dqcdRngX[nid][tid].s; i < dqcdRngX[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		//uxm
		QCDDopr_Load_Stride(hv,(pXM+i),(dqcdNy*dqcdNz*dqcdNt),DOPRSET_HSPIN);
		QCD_UXM_SET(v,hv,kappa);

		QCDDopr_Store_Stride(v,pV,dqcdNsite,DOPRSET_SPIN);

		pV += stride * dqcdNx;
	}
}


void QCDDopr_MakeXPB_EO(QCDHalfSpinor* pXP,QCDSpinor* pWP,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	// QCDComplex tv[6*QCD_NUM_SIMD];
	// QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,y,z,t,ipar;

	ieo += qcdBaseIeo;

	i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;
				if(ipar != 0){
					QCDDopr_Load(v,pWP,DOPRSET_SPIN);

					//for Plus boundary (send to minus)
					QCD_UXP_HALF(hv,v);

					QCDDopr_Store(hv,(pXP+i),DOPRSET_HSPIN);
					i++;
				}
				pWP += Nx;
			}
		}
	}
}

void QCDDopr_MakeXMB_EO(QCDHalfSpinor* pXM,QCDMatrix* pUM,QCDSpinor* pWM,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,y,z,t,ipar;

	ieo += qcdBaseIeo;

	i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;
				if(ipar == 0){
					QCDDopr_Load(v,pWM,DOPRSET_SPIN);
					QCDDopr_LoadGauge(u,pUM);

					//for Minus boundary (send to plus)
					QCD_UXM_HALF(tv,v);
					QCD_MUL_UM(hv,u,tv);

					QCDDopr_Store(hv,(pXM+i),DOPRSET_HSPIN);
					i++;
				}
				pWM += Nx;
				pUM += Nx;
			}
		}
	}
}

void QCDDopr_XPin_EO(QCDSpinor* pV,QCDMatrix* pUP,QCDSpinor* pWP,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,x,y,z,t,ipar,n,iw;
	int x,y,z,t,ipar,n,iw;

	ieo += qcdBaseIeo;

	// i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;

				if(ipar){
					n = Nx - 1;
					iw = 1;
				}
				else{
					n = Nx;
					iw = 0;
				}
				for(x=0;x<n;x++){
					QCDDopr_Load(v,pV + x,DOPRSET_SPIN);


					//uxp
					QCDDopr_Load(w,pWP + iw + x,DOPRSET_SPIN);
					QCDDopr_LoadGauge(u,pUP + x);

					QCD_UXP_HALF(tv,w);
					QCD_MUL_UP(hv,u,tv);
					QCD_UXP_SET(v,hv,kappa);

					QCDDopr_Store(v,pV + x,DOPRSET_SPIN);
				}
				pV += Nx;
				pWP += Nx;
				pUP += Nx;
			}
		}
	}
}

void QCDDopr_XMin_EO(QCDSpinor* pV,QCDMatrix* pUM,QCDSpinor* pWM,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,x,y,z,t,ipar,n,iv;
	int x,y,z,t,ipar,n,iv;

	ieo += qcdBaseIeo;

	// i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;

				if(ipar == 0){
					n = Nx - 1;
					iv = 1;
				}
				else{
					n = Nx;
					iv = 0;
				}
				for(x=0;x<n;x++){
					QCDDopr_Load(v,pV + x + iv,DOPRSET_SPIN);

					//uxm
					QCDDopr_Load(w,pWM + x,DOPRSET_SPIN);
					QCDDopr_LoadGauge(u,pUM + x);
					QCD_UXM_HALF(tv,w);
					QCD_MUL_UM(hv,u,tv);
					QCD_UXM_SET(v,hv,kappa);

					QCDDopr_Store(v,pV + x + iv,DOPRSET_SPIN);
				}
				pV += Nx;
				pWM += Nx;
				pUM += Nx;
			}
		}
	}
}

void QCDDopr_SetXPBnd_EO(QCDSpinor* pV,QCDMatrix* pUP,QCDHalfSpinor* pXP,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	// QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,x,y,z,t,ipar,n,iv;
	int i,y,z,t,ipar;

	ieo += qcdBaseIeo;

	pV += Nx - 1;
	pUP += Nx - 1;
	i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;
				if(ipar){
					QCDDopr_Load(v,pV,DOPRSET_SPIN);

					//uxp
					QCDDopr_Load(hv,(pXP + i),DOPRSET_HSPIN);
					QCDDopr_LoadGauge(u,pUP);

					QCD_MUL_UP(tv,u,hv);
					QCD_UXP_SET(v,tv,kappa);

					QCDDopr_Store(v,pV,DOPRSET_SPIN);
					i++;
				}
				pV += Nx;
				pUP += Nx;
			}
		}
	}
}


void QCDDopr_SetXMBnd_EO(QCDSpinor* pV,QCDHalfSpinor* pXM,int Nx,int Ny,int Nz,int Nt,int ieo,int tid,int nid)
{
	// QCDComplex tv[6*QCD_NUM_SIMD];
	// QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,x,y,z,t,ipar,n,iv;
	int i,y,z,t,ipar;

	ieo += qcdBaseIeo;

	i = 0;
	for(t=0;t<Nt;t++){
		for(z=0;z<Nz;z++){
			for(y=0;y<Ny;y++){
				ipar = (ieo + y + z + t) % 2;
				if(ipar == 0){
					QCDDopr_Load(v,pV,DOPRSET_SPIN);

					//uxm
					QCDDopr_Load(hv,(pXM+i),DOPRSET_HSPIN);
					QCD_UXM_SET(v,hv,kappa);

					QCDDopr_Store(v,pV,DOPRSET_SPIN);
					i++;
				}
				pV += Nx;
			}
		}
	}
}

__global__ void cuQCDDopr_MakeYPB(QCDComplex* pYP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pYP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNx;
	pWP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYInBnd[nid][tid].s; j < dqcdRngYInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pWP + j),dqcdNsite,DOPRSET_SPIN);

			//for Plus boundary (send to minus)
			QCD_UYP_HALF(hv,v);

#ifdef QCD_SHIFT_Y
			QCDDopr_StoreShiftMinus(hv,(pYP + j),DOPRSET_HSPIN);
#else
			QCDDopr_Store_Stride(hv,(pYP + j),(dqcdNx*dqcdNz*dqcdNt),DOPRSET_HSPIN);
#endif
		}
		pYP += gridDim.x * dqcdNx;
		pWP += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_MakeYMB(QCDComplex* pYM,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pYM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNx;
	pWM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pUM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYInBnd[nid][tid].s; j < dqcdRngYInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pWM + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUM + j));

			//for Minus boundary (send to plus)
			QCD_UYM_HALF(tv,v);
			QCD_MUL_UM(hv,u,tv);

#ifdef QCD_SHIFT_Y
			QCDDopr_StoreShiftPlus(hv,(pYM + j),DOPRSET_HSPIN);
#else
			QCDDopr_Store_Stride(hv,(pYM + j),(dqcdNx*dqcdNz*dqcdNt),DOPRSET_HSPIN);
#endif

		}
		pYM += gridDim.x * dqcdNx;
		pWM += gridDim.x * dqcdNxy;
		pUM += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_YPin(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pWP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pUP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pV += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYIn[nid][tid].s ; j < dqcdRngYIn[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uyp
			QCDDopr_Load_Stride(w,(pWP + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUP + j));

			QCD_UYP_HALF(tv,w);
			QCD_MUL_UP(hv,u,tv);
			QCD_UYP_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pV += gridDim.x * dqcdNxy;
		pUP += gridDim.x * dqcdNxy;
		pWP += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_YMin(QCDComplex* pV,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pV += dqcdNx;
	pUM += dqcdNx;
	pWM += dqcdNx;
	pWM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pUM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pV += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYIn[nid][tid].s; j < dqcdRngYIn[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uym
			QCDDopr_Load_Stride(w,(pWM + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUM + j));
			QCD_UYM_HALF(tv,w);
			QCD_MUL_UM(hv,u,tv);
			QCD_UYM_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pV += gridDim.x * dqcdNxy;
		pUM += gridDim.x * dqcdNxy;
		pWM += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_SetYPBnd(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pYP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pYP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNx;
	pV += dqcdNxy - dqcdNx;
	pUP += dqcdNxy - dqcdNx;
	pUP += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pV += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYInBnd[nid][tid].s; j < dqcdRngYInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uyp
			QCDDopr_Load_Stride(hv,(pYP + j),(dqcdNx*dqcdNz*dqcdNt),DOPRSET_HSPIN);
			QCDDopr_LoadGauge_Stride(u,(pUP + j));
			QCD_MUL_UP(tv,u,hv);
			QCD_UYP_SET(v,tv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pYP += gridDim.x * dqcdNx;
		pV += gridDim.x * dqcdNxy;
		pUP += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_SetYMBnd(QCDComplex* pV,QCDComplex* pYM,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pYM += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNx;
	pV += (dqcdRngYOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	for(i = blockIdx.x + dqcdRngYOut[nid][tid].s; i < dqcdRngYOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngYInBnd[nid][tid].s; j < dqcdRngYInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uym
			QCDDopr_Load_Stride(hv,(pYM+j),(dqcdNx*dqcdNz*dqcdNt),DOPRSET_HSPIN);
			QCD_UYM_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pYM += gridDim.x * dqcdNx;
		pV += gridDim.x * dqcdNxy;
	}
}

__global__ void cuQCDDopr_MakeZPB(QCDComplex* pZP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pZP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pWP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngZInBnd[nid][tid].s; j < dqcdRngZInBnd[nid][tid].e; j += blockDim.x){
			//for Plus boundary (send to minus)
		        QCDDopr_Load_Stride(v,(pWP + j),dqcdNsite,DOPRSET_SPIN);
			QCD_UZP_HALF(hv,v);

#ifdef QCD_SHIFT_Z
			QCDDopr_StoreShiftMinus(hv,(pZP + j),DOPRSET_HSPIN);
#else
			QCDDopr_Store_Stride(hv,(pZP + j),(dqcdNx*dqcdNy*dqcdNt),DOPRSET_HSPIN);
#endif
		}
		pZP += gridDim.x * dqcdNxy;
		pWP += gridDim.x * dqcdNxyz;
	}
}

__global__ void cuQCDDopr_MakeZMB(QCDComplex* pZM,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pZM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pWM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pUM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngZInBnd[nid][tid].s; j < dqcdRngZInBnd[nid][tid].e; j += blockDim.x){
			//for Minus boundary (send to plus)
		        QCDDopr_Load_Stride(v,(pWM + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUM + j));

			QCD_UZM_HALF(tv,v);
			QCD_MUL_UM(hv,u,tv);

#ifdef QCD_SHIFT_Z
			QCDDopr_StoreShiftPlus(hv,(pZM + j),DOPRSET_HSPIN);
#else
			QCDDopr_Store_Stride(hv,(pZM + j),(dqcdNx*dqcdNy*dqcdNt),DOPRSET_HSPIN);
#endif

		}
		pZM += gridDim.x * dqcdNxy;
		pWM += gridDim.x * dqcdNxyz;
		pUM += gridDim.x * dqcdNxyz;
	}
}

__global__ void cuQCDDopr_ZPin(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pV += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pWP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pUP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
	        for(j = threadIdx.x + dqcdRngZIn[nid][tid].s; j < dqcdRngZIn[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uzp
			QCDDopr_Load_Stride(w,(pWP + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUP + j));

			QCD_UZP_HALF(tv,w);
			QCD_MUL_UP(hv,u,tv);
			QCD_UZP_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pV += gridDim.x * dqcdNxyz;
		pUP += gridDim.x * dqcdNxyz;
		pWP += gridDim.x * dqcdNxyz;
	}
}

__global__ void cuQCDDopr_ZMin(QCDComplex* pV,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pV += dqcdNxy;
	pUM += dqcdNxy;
	pWM += dqcdNxy;
	pV += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pWM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pUM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
	        for(j = threadIdx.x + dqcdRngZIn[nid][tid].s; j < dqcdRngZIn[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uzm
			QCDDopr_Load_Stride(w,(pWM + j),dqcdNsite,DOPRSET_SPIN);
			QCDDopr_LoadGauge_Stride(u,(pUM + j));
			QCD_UZM_HALF(tv,w);
			QCD_MUL_UM(hv,u,tv);
			QCD_UZM_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pV += gridDim.x * dqcdNxyz;
		pUM += gridDim.x * dqcdNxyz;
		pWM += gridDim.x * dqcdNxyz;
	}
}

__global__ void cuQCDDopr_SetZPBnd(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pZP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pV += dqcdNxyz - dqcdNxy;
	pUP += dqcdNxyz - dqcdNxy;

	pZP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pUP += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	pV += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
		for(j = threadIdx.x + dqcdRngZInBnd[nid][tid].s; j < dqcdRngZInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);

			//uzp
			QCDDopr_Load_Stride(hv,(pZP + j),(dqcdNx*dqcdNy*dqcdNt),DOPRSET_HSPIN);
			QCDDopr_LoadGauge_Stride(u,(pUP + j));
			QCD_MUL_UP(tv,u,hv);
			QCD_UZP_SET(v,tv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pZP += gridDim.x * dqcdNxy;
		pUP += gridDim.x * dqcdNxyz;
		pV += gridDim.x * dqcdNxyz;
	}
}

__global__ void cuQCDDopr_SetZMBnd(QCDComplex* pV,QCDComplex* pZM,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i,j;
	// int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	// int stride = blockDim.x * gridDim.x;

	pZM += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxy;
	pV += (dqcdRngZOut[nid][tid].s + blockIdx.x) * dqcdNxyz;
	for(i = blockIdx.x + dqcdRngZOut[nid][tid].s; i < dqcdRngZOut[nid][tid].e; i += gridDim.x){
		for(j= threadIdx.x + dqcdRngZInBnd[nid][tid].s; j < dqcdRngZInBnd[nid][tid].e; j += blockDim.x){
		        QCDDopr_Load_Stride(v,(pV+j),dqcdNsite,DOPRSET_SPIN);

			//uzm
			QCDDopr_Load_Stride(hv,(pZM+j),(dqcdNx*dqcdNy*dqcdNt),DOPRSET_HSPIN);
			QCD_UZM_SET(v,hv,kappa);

			QCDDopr_Store_Stride(v,(pV + j),dqcdNsite,DOPRSET_SPIN);
		}
		pZM += gridDim.x * dqcdNxy;
		pV += gridDim.x * dqcdNxyz;
	}
}

void QCDDopr_MakeTPB(QCDHalfSpinor* pTP,QCDSpinor* pWP,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;

	for(i=qcdRngTBnd[nid][tid].s;i<qcdRngTBnd[nid][tid].e;i++){
		//for Plus boundary (send to minus)
		QCDDopr_Load(v,(pWP + j),DOPRSET_SPIN);

		QCD_UTP_HALF(hv,v);

		QCDDopr_Store(hv,(pTP + j),DOPRSET_HSPIN);
	}
}

void QCDDopr_MakeTMB(QCDHalfSpinor* pTM,QCDMatrix* pUM,QCDSpinor* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i,j;

	for(i=qcdRngTBnd[nid][tid].s;i<qcdRngTBnd[nid][tid].e;i++){
		//for Minus boundary (send to plus)
		QCDDopr_Load(v,(pWM + j),DOPRSET_SPIN);
		QCDDopr_LoadGauge(u,(pUM + j));

		QCD_UTM_HALF(tv,v);
		QCD_MUL_UM(hv,u,tv);

		QCDDopr_Store(hv,(pTM + j),DOPRSET_HSPIN);
	}
}

void QCDDopr_TPin(QCDSpinor* pV,QCDMatrix* pUP,QCDSpinor* pWP,int Nin,int Nt,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,j,t;
	int j,t;

	for(t=0;t<Nt-1;t++){
		for(j=0;j<Nin;j++){
			QCDDopr_Load(v,pV,DOPRSET_SPIN);

			//utp
			QCDDopr_Load(w,pWP,DOPRSET_SPIN);
			QCDDopr_LoadGauge(u,pUP);
			QCD_UTP_HALF(tv,w);
			QCD_MUL_UP(hv,u,tv);
			QCD_UTP_SET(v,hv,kappa);

			QCDDopr_Store(v,pV,DOPRSET_SPIN);

			pV += 1;
			pUP += 1;
			pWP += 1;
		}
	}
}

void QCDDopr_TMin(QCDSpinor* pV,QCDMatrix* pUM,QCDSpinor* pWM,int Nin,int Nt,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,j,t;
	int j,t;

	pV += Nin;
	pUM += Nin;
	pWM += Nin;

	for(t=1;t<Nt;t++){
		for(j=0;j<Nin;j++){
			QCDDopr_Load(v,pV,DOPRSET_SPIN);

			//utm
			QCDDopr_Load(w,pWM,DOPRSET_SPIN);
			QCDDopr_LoadGauge(u,pUM);
			QCD_UTM_HALF(tv,w);
			QCD_MUL_UM(hv,u,tv);
			QCD_UTM_SET(v,hv,kappa);

			QCDDopr_Store(v,pV,DOPRSET_SPIN);

			pV += 1;
			pUM += 1;
			pWM += 1;
		}
	}

}


void QCDDopr_SetTPBnd(QCDSpinor* pV,QCDMatrix* pUP,QCDHalfSpinor* pTP,int Nin,int Nt,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	// QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,j,t;
	int j;

	pV += Nin*Nt - Nin;
	pUP += Nin*Nt - Nin;

	for(j=0;j<Nin;j++){
		QCDDopr_Load(v,pV,DOPRSET_SPIN);

		//utp
		QCDDopr_Load(hv,(pTP + j),DOPRSET_HSPIN);
		QCDDopr_LoadGauge(u,pUP);
		QCD_MUL_UP(tv,u,hv);
		QCD_UTP_SET(v,tv,kappa);

		QCDDopr_Store(v,pV,DOPRSET_SPIN);

		pV += 1;
		pUP += 1;
	}
}


void QCDDopr_SetTMBnd(QCDSpinor* pV,QCDHalfSpinor* pTM,int Nin,int Nt,int tid,int nid)
{
	// QCDComplex tv[6*QCD_NUM_SIMD];
	// QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = qcdtKappa;
	// int i,j,t;
	int j;

	for(j=0;j<Nin;j++){
		QCDDopr_Load(v,pV,DOPRSET_SPIN);

		//utm
		QCDDopr_Load(hv,(pTM+j),DOPRSET_HSPIN);
		QCD_UTM_SET(v,hv,kappa);

		QCDDopr_Store(v,pV,DOPRSET_SPIN);

		pV += 1;
	}
}

__global__ void cuQCDDopr_MakeTPB_dirac(QCDComplex* pTP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(i = gtid + dqcdRngTBnd[nid][tid].s; i < dqcdRngTBnd[nid][tid].e; i += stride){
		//for Plus boundary (send to minus)
	        QCDDopr_Load_Stride(v,(pWP + i),dqcdNsite,DOPRSET_SPIN);

		QCD_UTP_DIRAC_HALF(hv,v);

		QCDDopr_Store_Stride(hv,(pTP + i),(dqcdNx*dqcdNy*dqcdNz),DOPRSET_HSPIN);
	}
}

__global__ void cuQCDDopr_MakeTMB_dirac(QCDComplex* pTM,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(i = gtid + dqcdRngTBnd[nid][tid].s; i < dqcdRngTBnd[nid][tid].e; i += stride){
		//for Minus boundary (send to plus)
	        QCDDopr_Load_Stride(v,(pWM + i),dqcdNsite,DOPRSET_SPIN);
		QCDDopr_LoadGauge_Stride(u,(pUM + i));

		QCD_UTM_DIRAC_HALF(tv,v);
		QCD_MUL_UM(hv,u,tv);

		QCDDopr_Store_Stride(hv,(pTM + i),(dqcdNx*dqcdNy*dqcdNz),DOPRSET_HSPIN);
	}
}

__global__ void cuQCDDopr_TPin_dirac(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pWP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	// int i,j,t;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(i = gtid + dqcdRngT[nid][tid].s; i < dqcdRngT[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);

		//utp
		QCDDopr_Load_Stride(w,(pWP + i),dqcdNsite,DOPRSET_SPIN);
		QCDDopr_LoadGauge_Stride(u,(pUP + i));
		QCD_UTP_DIRAC_HALF(tv,w);
		QCD_MUL_UP(hv,u,tv);
		QCD_UTP_DIRAC_SET(v,hv,kappa);

		QCDDopr_Store_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);

	}
}

__global__ void cuQCDDopr_TMin_dirac(QCDComplex* pV,QCDComplex* pUM,QCDComplex* pWM,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],w[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV += dqcdNxyz;
	pUM += dqcdNxyz;
	pWM += dqcdNxyz;

	for(i = gtid + dqcdRngT[nid][tid].s; i < dqcdRngT[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);

		//utm
		QCDDopr_Load_Stride(w,(pWM + i),dqcdNsite,DOPRSET_SPIN);
		QCDDopr_LoadGauge_Stride(u,(pUM + i));
		QCD_UTM_DIRAC_HALF(tv,w);
		QCD_MUL_UM(hv,u,tv);
		QCD_UTM_DIRAC_SET(v,hv,kappa);

		QCDDopr_Store_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);
	}

}

__global__ void cuQCDDopr_SetTPBnd_dirac(QCDComplex* pV,QCDComplex* pUP,QCDComplex* pTP,int tid,int nid)
{
	QCDComplex tv[6*QCD_NUM_SIMD];
	QCDComplex u[9*QCD_NUM_SIMD],v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV += dqcdNsite - dqcdNxyz;
	pUP += dqcdNsite - dqcdNxyz;

	for(i = gtid + dqcdRngTBnd[nid][tid].s; i < dqcdRngTBnd[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);

		//utp
		QCDDopr_Load_Stride(hv,(pTP + i),(dqcdNx*dqcdNy*dqcdNz),DOPRSET_HSPIN);
		QCDDopr_LoadGauge_Stride(u,(pUP + i));
		QCD_MUL_UP(tv,u,hv);
		QCD_UTP_DIRAC_SET(v,tv,kappa);

		QCDDopr_Store_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);
	}
}


__global__ void cuQCDDopr_SetTMBnd_dirac(QCDComplex* pV,QCDComplex* pTM,int tid,int nid)
{
	QCDComplex v[12*QCD_NUM_SIMD],hv[6*QCD_NUM_SIMD];
	QCDReal* kappa = dqcdtKappa;
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	for(i = gtid + dqcdRngTBnd[nid][tid].s; i < dqcdRngTBnd[nid][tid].e; i += stride){
	        QCDDopr_Load_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);

		//utm
		QCDDopr_Load_Stride(hv,(pTM+i),(dqcdNx*dqcdNy*dqcdNz),DOPRSET_HSPIN);
		QCD_UTM_DIRAC_SET(v,hv,kappa);

		QCDDopr_Store_Stride(v,(pV + i),dqcdNsite,DOPRSET_SPIN);
	}
}


#define QCDDopr_GetRank(x,y,z,t) \
	((x) + (y) * qcdNetSize[0] + (z) * qcdNetSize[0]*qcdNetSize[1] + (t) * qcdNetSize[0]*qcdNetSize[1]*qcdNetSize[2])


int QCDGetGCD(int a,int b)
{
	int i,j,t;

	if(a == b){
		return a;
	}
	else if(a > b){
		t = b;
		j = a;
	}
	else{
		j = b;
		t = a;
	}
	do{
		i = j;
		j = t;
		t = j % i;
	}while(t != 0);

	return i;
}


void QCDDopr_Init(int Nx,int Ny,int Nz,int Nt,int npx,int npy,int npz,int npt,int myrank)
{
	int tid,nid;
	int i,noY,niY,noZ,niZ,ii,io;

	qcdNProcs = npx*npy*npz*npt;
	qcdMyRank = myrank;
	qcdNx = Nx;
	qcdNy = Ny;
	qcdNz = Nz;
	qcdNt = Nt;
	qcdNxy = Nx*Ny;
	qcdNxyz = qcdNxy*Nz;
	qcdNsite = qcdNxyz*Nt;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNx), &qcdNx, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNy), &qcdNy, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNz), &qcdNz, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNt), &qcdNt, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNxy), &qcdNxy, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNxyz), &qcdNxyz, sizeof(int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdNsite), &qcdNsite, sizeof(int)));

	for (i = 0; i < 8; i++) {
	// for (i = 0; i < 9; i++) {
	    checkCudaErrors(hipStreamCreateWithFlags(&stream[i], hipStreamNonBlocking));
	    // checkCudaErrors(hipStreamCreate(&stream[i]));
	    // checkCudaErrors(hipEventCreateWithFlags(&event[i], hipEventDisableTiming));
	}

	qcdNetSize[0] = npx;
	qcdNetSize[1] = npy;
	qcdNetSize[2] = npz;
	qcdNetSize[3] = npt;

	qcdNetPos[0] = qcdMyRank % qcdNetSize[0];
	qcdNetPos[1] = (qcdMyRank/qcdNetSize[0]) % qcdNetSize[1];
	qcdNetPos[2] = (qcdMyRank/(qcdNetSize[0]*qcdNetSize[1])) % qcdNetSize[2];
	qcdNetPos[3] = qcdMyRank/(qcdNetSize[0]*qcdNetSize[1]*qcdNetSize[2]);

	qcdRankNeighbors[QCD_XP] = QCDDopr_GetRank((qcdNetPos[0] + 1) % qcdNetSize[0],qcdNetPos[1],qcdNetPos[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_XM] = QCDDopr_GetRank((qcdNetPos[0] + qcdNetSize[0] - 1) % qcdNetSize[0],qcdNetPos[1],qcdNetPos[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_YP] = QCDDopr_GetRank(qcdNetPos[0],(qcdNetPos[1] + 1) % qcdNetSize[1],qcdNetPos[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_YM] = QCDDopr_GetRank(qcdNetPos[0],(qcdNetPos[1] + qcdNetSize[1] - 1) % qcdNetSize[1],qcdNetPos[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_ZP] = QCDDopr_GetRank(qcdNetPos[0],qcdNetPos[1],(qcdNetPos[2] + 1) % qcdNetSize[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_ZM] = QCDDopr_GetRank(qcdNetPos[0],qcdNetPos[1],(qcdNetPos[2] + qcdNetSize[2] - 1) % qcdNetSize[2],qcdNetPos[3]);
	qcdRankNeighbors[QCD_TP] = QCDDopr_GetRank(qcdNetPos[0],qcdNetPos[1],qcdNetPos[2],(qcdNetPos[3] + 1) % qcdNetSize[3]);
	qcdRankNeighbors[QCD_TM] = QCDDopr_GetRank(qcdNetPos[0],qcdNetPos[1],qcdNetPos[2],(qcdNetPos[3] + qcdNetSize[3] - 1) % qcdNetSize[3]);


        qcdSendBuf[QCD_XP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt);
        qcdSendBuf[QCD_XM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt);
        qcdRecvBuf[QCD_XP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt);
        qcdRecvBuf[QCD_XM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt);

	qcdSendBuf[QCD_YP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt);
        qcdSendBuf[QCD_YM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt);
	qcdRecvBuf[QCD_YP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt);
        qcdRecvBuf[QCD_YM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt);

        qcdSendBuf[QCD_ZP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt);
	qcdSendBuf[QCD_ZM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt);
        qcdRecvBuf[QCD_ZP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt);
	qcdRecvBuf[QCD_ZM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt);

        qcdSendBuf[QCD_TP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz);
        qcdSendBuf[QCD_TM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz);
        qcdRecvBuf[QCD_TP] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz);
        qcdRecvBuf[QCD_TM] = (QCDComplex*)malloc(sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz);

	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));

	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));

	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));

	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipHostMalloc((void**)&pqcdSendBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipHostMalloc((void**)&pqcdRecvBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));

	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt));

	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt));

	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt));

	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipMalloc((void**)&dqcdSendBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));
	checkCudaErrors(hipMalloc((void**)&dqcdRecvBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz));


	for(nid=1;nid<QCD_NUM_MAX_THREADS;nid++){
		//y
		noY = QCDGetGCD(qcdNt*qcdNz,nid);
		niY = nid / noY;

		//z
		noZ = QCDGetGCD(qcdNt,nid);
		niZ = nid / noZ;

		for(tid=0;tid<nid;tid++){
			//x
			qcdRngX[nid][tid].s = tid * qcdNy*qcdNz*qcdNt / nid;
			qcdRngX[nid][tid].e = (tid + 1) * qcdNy*qcdNz*qcdNt / nid;

			//y
			ii = tid % niY;
			io = tid / niY;
			qcdRngYOut[nid][tid].s = io * qcdNt*qcdNz / noY;
			qcdRngYOut[nid][tid].e = (io + 1) * qcdNt*qcdNz / noY;

			qcdRngYInBnd[nid][tid].s = ii * qcdNx / niY;
			qcdRngYInBnd[nid][tid].e = (ii + 1) * qcdNx / niY;

			qcdRngYIn[nid][tid].s = ii * (qcdNxy - qcdNx) / niY;
			qcdRngYIn[nid][tid].e = (ii + 1) * (qcdNxy - qcdNx) / niY;

			//z
			ii = tid % niZ;
			io = tid / niZ;

			qcdRngZOut[nid][tid].s = io * qcdNt / noZ;
			qcdRngZOut[nid][tid].e = (io + 1) * qcdNt / noZ;
			qcdRngZInBnd[nid][tid].s = ii * qcdNxy / niZ;
			qcdRngZInBnd[nid][tid].e = (ii + 1) * qcdNxy / niZ;
			qcdRngZIn[nid][tid].s = ii * (qcdNxyz-qcdNxy) / niZ;
			qcdRngZIn[nid][tid].e = (ii + 1) * (qcdNxyz-qcdNxy) / niZ;

			//t
			qcdRngT[nid][tid].s = tid * (qcdNsite - qcdNxyz) / nid;
			qcdRngT[nid][tid].e = (tid + 1) * (qcdNsite - qcdNxyz) / nid;

			qcdRngTBnd[nid][tid].s = tid * (qcdNxyz) / nid;
			qcdRngTBnd[nid][tid].e = (tid + 1) * (qcdNxyz) / nid;
		}
	}

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngX), qcdRngX, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngYOut), qcdRngYOut, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngYIn), qcdRngYIn, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngYInBnd), qcdRngYInBnd, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngZOut), qcdRngZOut, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngZIn), qcdRngZIn, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngZInBnd), qcdRngZInBnd, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngT), qcdRngT, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdRngTBnd), qcdRngTBnd, sizeof(QCDRng)*QCD_NUM_MAX_THREADS*QCD_NUM_MAX_THREADS));
}

void cuQCDDopr_Mult(QCDComplex* dpV,QCDComplex* dpU,QCDComplex* dpW,double k)
{
	MPI_Request reqSend[8];
	MPI_Request reqRecv[8];
	MPI_Status st;
	// int i;

	QCDComplex* dpUx;
	QCDComplex* dpUy;
	QCDComplex* dpUz;
	QCDComplex* dpUt;

	dim3 threads(128,1,1);
	dim3 blocks(qcdNsite/threads.x,1,1);

	qcdtKappa[0] = k;
	qcdtKappa[1] = k;
	qcdtKappa[2] = k;
	qcdtKappa[3] = k;

	//debug
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dqcdtKappa), qcdtKappa, sizeof(QCDReal)*4));

	dpUx = dpU;
	dpUy = dpU + qcdNsite;
	dpUz = dpU + qcdNsite*2;
	dpUt = dpU + qcdNsite*3;

/* #pragma omp parallel num_threads(8) */
#pragma omp parallel
	{
	int tid = 0,nid = 1;

#if 0
	tid = omp_get_thread_num();
	nid = omp_get_num_threads();
#endif
	/* //debug */
	// printf("nthreads: %d\n", nid);
	// printf("max_threads: %d\n", omp_get_max_threads());

	if(tid == 0){
		MPI_Irecv(qcdRecvBuf[QCD_TP],12*qcdNxyz,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_TP],QCD_TP,MPI_COMM_WORLD,&reqRecv[QCD_TP]);
		MPI_Irecv(qcdRecvBuf[QCD_TM],12*qcdNxyz,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_TM],QCD_TM,MPI_COMM_WORLD,&reqRecv[QCD_TM]);

		MPI_Irecv(qcdRecvBuf[QCD_XP],12*qcdNy*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_XP],QCD_XP,MPI_COMM_WORLD,&reqRecv[QCD_XP]);
		MPI_Irecv(qcdRecvBuf[QCD_XM],12*qcdNy*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_XM],QCD_XM,MPI_COMM_WORLD,&reqRecv[QCD_XM]);

		MPI_Irecv(qcdRecvBuf[QCD_YP],12*qcdNx*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_YP],QCD_YP,MPI_COMM_WORLD,&reqRecv[QCD_YP]);
		MPI_Irecv(qcdRecvBuf[QCD_YM],12*qcdNx*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_YM],QCD_YM,MPI_COMM_WORLD,&reqRecv[QCD_YM]);

		MPI_Irecv(qcdRecvBuf[QCD_ZP],12*qcdNx*qcdNy*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_ZP],QCD_ZP,MPI_COMM_WORLD,&reqRecv[QCD_ZP]);
		MPI_Irecv(qcdRecvBuf[QCD_ZM],12*qcdNx*qcdNy*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_ZM],QCD_ZM,MPI_COMM_WORLD,&reqRecv[QCD_ZM]);

	}

	cuQCDDopr_MakeTPB_dirac<<<blocks, threads, 0, stream[QCD_TP]>>>(dqcdSendBuf[QCD_TP],dpW,tid,nid);
	cuQCDDopr_MakeTMB_dirac<<<blocks, threads, 0, stream[QCD_TM]>>>(dqcdSendBuf[QCD_TM],dpUt + qcdNsite-qcdNxyz,dpW + qcdNsite-qcdNxyz,tid,nid);
	cuQCDDopr_MakeXPB<<<blocks, threads, 0, stream[QCD_XP]>>>(dqcdSendBuf[QCD_XP],dpW,tid,nid);
	cuQCDDopr_MakeXMB<<<blocks, threads, 0, stream[QCD_XM]>>>(dqcdSendBuf[QCD_XM],dpUx + qcdNx-1,dpW + qcdNx-1,tid,nid);
	cuQCDDopr_MakeYPB<<<blocks, threads, 0, stream[QCD_YP]>>>(dqcdSendBuf[QCD_YP],dpW,tid,nid);
	cuQCDDopr_MakeYMB<<<blocks, threads, 0, stream[QCD_YM]>>>(dqcdSendBuf[QCD_YM],dpUy + qcdNxy-qcdNx,dpW + qcdNxy-qcdNx,tid,nid);
	cuQCDDopr_MakeZPB<<<blocks, threads, 0, stream[QCD_ZP]>>>(dqcdSendBuf[QCD_ZP],dpW,tid,nid);
	cuQCDDopr_MakeZMB<<<blocks, threads, 0, stream[QCD_ZM]>>>(dqcdSendBuf[QCD_ZM],dpUz + qcdNxyz-qcdNxy,dpW + qcdNxyz-qcdNxy,tid,nid);

	//Send T
#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_TP], dqcdSendBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyDeviceToHost, stream[QCD_TP]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_TP], pqcdSendBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToHost, stream[QCD_TP]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_TP]));
	    MPI_Isend(qcdSendBuf[QCD_TP],12*qcdNxyz,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_TM],QCD_TP,MPI_COMM_WORLD,&reqSend[QCD_TP]);
	}

#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_TM], dqcdSendBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyDeviceToHost, stream[QCD_TM]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_TM], pqcdSendBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToHost, stream[QCD_TM]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_TM]));
	    MPI_Isend(qcdSendBuf[QCD_TM],12*qcdNxyz,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_TP],QCD_TM,MPI_COMM_WORLD,&reqSend[QCD_TM]);
	}

	//Send X
#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_XP], dqcdSendBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyDeviceToHost, stream[QCD_XP]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_XP], pqcdSendBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_XP]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_XP]));
	    MPI_Isend(qcdSendBuf[QCD_XP],12*qcdNy*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_XM],QCD_XP,MPI_COMM_WORLD,&reqSend[QCD_XP]);
	}

#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_XM], dqcdSendBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyDeviceToHost, stream[QCD_XM]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_XM], pqcdSendBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_XM]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_XM]));
	    MPI_Isend(qcdSendBuf[QCD_XM],12*qcdNy*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_XP],QCD_XM,MPI_COMM_WORLD,&reqSend[QCD_XM]);
	}

	//Send Y
#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_YP], dqcdSendBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyDeviceToHost, stream[QCD_YP]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_YP], pqcdSendBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_YP]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_YP]));
	    MPI_Isend(qcdSendBuf[QCD_YP],12*qcdNx*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_YM],QCD_YP,MPI_COMM_WORLD,&reqSend[QCD_YP]);
	}

#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_YM], dqcdSendBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyDeviceToHost, stream[QCD_YM]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_YM], pqcdSendBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_YM]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_YM]));
	    MPI_Isend(qcdSendBuf[QCD_YM],12*qcdNx*qcdNz*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_YP],QCD_YM,MPI_COMM_WORLD,&reqSend[QCD_YM]);
	}

	//Send Z
#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_ZP], dqcdSendBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyDeviceToHost, stream[QCD_ZP]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_ZP], pqcdSendBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToHost, stream[QCD_ZP]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_ZP]));
	    MPI_Isend(qcdSendBuf[QCD_ZP],12*qcdNx*qcdNy*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_ZM],QCD_ZP,MPI_COMM_WORLD,&reqSend[QCD_ZP]);
	}

#pragma omp barrier
	if(tid == 0){
	    checkCudaErrors(hipMemcpyAsync(pqcdSendBuf[QCD_ZM], dqcdSendBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyDeviceToHost, stream[QCD_ZM]));
	    checkCudaErrors(hipMemcpyAsync(qcdSendBuf[QCD_ZM], pqcdSendBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToHost, stream[QCD_ZM]));
	    checkCudaErrors(hipStreamSynchronize(stream[QCD_ZM]));
	    MPI_Isend(qcdSendBuf[QCD_ZM],12*qcdNx*qcdNy*qcdNt,MPI_DOUBLE_PRECISION,qcdRankNeighbors[QCD_ZP],QCD_ZM,MPI_COMM_WORLD,&reqSend[QCD_ZM]);
	}

	cuQCDLA_Equate<<<blocks, threads>>>(dpV + tid*qcdNsite/nid,dpW + tid*qcdNsite/nid, (tid+1)*qcdNsite/nid - tid*qcdNsite/nid);
	cuQCDDopr_TPin_dirac<<<blocks, threads>>>(dpV,dpUt,dpW + qcdNxyz,tid,nid);
	cuQCDDopr_TMin_dirac<<<blocks, threads>>>(dpV,dpUt-qcdNxyz,dpW - qcdNxyz,tid,nid);
	cuQCDDopr_XPin<<<blocks, threads>>>(dpV,dpUx,dpW+1,tid,nid);
	cuQCDDopr_XMin<<<blocks, threads>>>(dpV,dpUx-1,dpW-1,tid,nid);
	cuQCDDopr_YPin<<<blocks, threads>>>(dpV,dpUy,dpW + qcdNx,tid,nid);
	cuQCDDopr_YMin<<<blocks, threads>>>(dpV,dpUy-qcdNx,dpW - qcdNx,tid,nid);
	cuQCDDopr_ZPin<<<blocks, threads>>>(dpV,dpUz,dpW + qcdNxy,tid,nid);
	cuQCDDopr_ZMin<<<blocks, threads>>>(dpV,dpUz-qcdNxy,dpW - qcdNxy,tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_TP],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_TP], qcdRecvBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToHost, stream[QCD_TP]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_TP], pqcdRecvBuf[QCD_TP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToDevice, stream[QCD_TP]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_TP]));
	cuQCDDopr_SetTPBnd_dirac<<<blocks, threads>>>(dpV,dpUt,dqcdRecvBuf[QCD_TP],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_TM],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_TM], qcdRecvBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToHost, stream[QCD_TM]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_TM], pqcdRecvBuf[QCD_TM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNz, hipMemcpyHostToDevice, stream[QCD_TM]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_TM]));
	cuQCDDopr_SetTMBnd_dirac<<<blocks, threads>>>(dpV,dqcdRecvBuf[QCD_TM],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_XP],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_XP], qcdRecvBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_XP]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_XP], pqcdRecvBuf[QCD_XP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToDevice, stream[QCD_XP]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_XP]));
	cuQCDDopr_SetXPBnd<<<blocks, threads>>>(dpV,dpUx,dqcdRecvBuf[QCD_XP],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_XM],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_XM], qcdRecvBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_XM]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_XM], pqcdRecvBuf[QCD_XM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNy*qcdNz*qcdNt, hipMemcpyHostToDevice, stream[QCD_XM]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_XM]));
	cuQCDDopr_SetXMBnd<<<blocks, threads>>>(dpV,dqcdRecvBuf[QCD_XM],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_YP],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_YP], qcdRecvBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_YP]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_YP], pqcdRecvBuf[QCD_YP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToDevice, stream[QCD_YP]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_YP]));
	cuQCDDopr_SetYPBnd<<<blocks, threads>>>(dpV,dpUy,dqcdRecvBuf[QCD_YP],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_YM],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_YM], qcdRecvBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToHost, stream[QCD_YM]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_YM], pqcdRecvBuf[QCD_YM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNz*qcdNt, hipMemcpyHostToDevice, stream[QCD_YM]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_YM]));
	cuQCDDopr_SetYMBnd<<<blocks, threads>>>(dpV,dqcdRecvBuf[QCD_YM],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_ZP],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_ZP], qcdRecvBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToHost, stream[QCD_ZP]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_ZP], pqcdRecvBuf[QCD_ZP], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToDevice, stream[QCD_ZP]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_ZP]));
	cuQCDDopr_SetZPBnd<<<blocks, threads>>>(dpV,dpUz,dqcdRecvBuf[QCD_ZP],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqRecv[QCD_ZM],&st);
		checkCudaErrors(hipMemcpyAsync(pqcdRecvBuf[QCD_ZM], qcdRecvBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToHost, stream[QCD_ZM]));
		checkCudaErrors(hipMemcpyAsync(dqcdRecvBuf[QCD_ZM], pqcdRecvBuf[QCD_ZM], sizeof(QCDComplex)*QCD_HALF_SPINOR_SIZE*qcdNx*qcdNy*qcdNt, hipMemcpyHostToDevice, stream[QCD_ZM]));
	}
#pragma omp barrier
	checkCudaErrors(hipStreamSynchronize(stream[QCD_ZM]));
	cuQCDDopr_SetZMBnd<<<blocks, threads>>>(dpV,dqcdRecvBuf[QCD_ZM],tid,nid);

	if(tid == 0){
		MPI_Wait(&reqSend[QCD_TP],&st);
		MPI_Wait(&reqSend[QCD_TM],&st);
		MPI_Wait(&reqSend[QCD_XP],&st);
		MPI_Wait(&reqSend[QCD_XM],&st);
		MPI_Wait(&reqSend[QCD_YP],&st);
		MPI_Wait(&reqSend[QCD_YM],&st);
		MPI_Wait(&reqSend[QCD_ZP],&st);
		MPI_Wait(&reqSend[QCD_ZM],&st);
	}
#pragma omp barrier

	}
}

__global__ void cuQCDopr_MultGamma5(QCDComplex* pV)
{
	int i;
	int gtid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	hipDoubleComplex v0,v1;

	for(i = gtid; i < dqcdNsite; i += stride){
	        v0 = pV[SPINOR_ID(i, 0)];
		v1 = pV[SPINOR_ID(i, 3)];
		pV[SPINOR_ID(i, 0)] = pV[SPINOR_ID(i, 6)];
		pV[SPINOR_ID(i, 3)] = pV[SPINOR_ID(i, 9)];
		pV[SPINOR_ID(i, 6)] = v0;
		pV[SPINOR_ID(i, 9)] = v1;

		v0 = pV[SPINOR_ID(i, 1)];
		v1 = pV[SPINOR_ID(i, 4)];
		pV[SPINOR_ID(i, 1)] = pV[SPINOR_ID(i, 7)];
		pV[SPINOR_ID(i, 4)] = pV[SPINOR_ID(i, 10)];
		pV[SPINOR_ID(i, 7)] = v0;
		pV[SPINOR_ID(i, 10)] = v1;

		v0 = pV[SPINOR_ID(i, 2)];
		v1 = pV[SPINOR_ID(i, 5)];
		pV[SPINOR_ID(i, 2)] = pV[SPINOR_ID(i, 8)];
		pV[SPINOR_ID(i, 5)] = pV[SPINOR_ID(i, 11)];
		pV[SPINOR_ID(i, 8)] = v0;
		pV[SPINOR_ID(i, 11)] = v1;
	}
}

void cuQCDDopr_H(QCDComplex* dpV,QCDComplex* dpU,QCDComplex* dpW,double k)
{
    dim3 threads(128,1,1);
    dim3 blocks(qcdNsite/threads.x,1,1);

    cuQCDDopr_Mult(dpV,dpU,dpW,-k);
    cuQCDopr_MultGamma5<<<blocks, threads>>>(dpV);

}

void cuQCDDopr_DdagD(QCDComplex* dpV,QCDComplex* dpU,QCDComplex* dpW,QCDComplex* dpT,double k)
{
    cuQCDDopr_H(dpT,dpU,dpW,k);
    cuQCDDopr_H(dpV,dpU,dpT,k);
}
