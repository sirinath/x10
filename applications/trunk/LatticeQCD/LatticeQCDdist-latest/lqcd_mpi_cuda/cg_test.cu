#include <stdio.h>
#include <stdlib.h>
// #include <complex.h>
#include <hip/hip_complex.h>

#include <sys/time.h>

#include <mpi.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "dslash_base.h"
#include "lib_vec.h"
#include "qcd.h"
#include "qcd_mult.h"


#define MYRAND_MAX 		32767
static unsigned long myrand_next;

static int myrand(void) 
{
	myrand_next = myrand_next * 1103515245 + 12345;
	return((unsigned)(myrand_next/65536) % 32768);
}

static void mysrand(unsigned seed) 
{
	myrand_next = seed;
}

double mysecond()
{
	struct timeval tp;
	struct timezone tzp;
	// int i;

	// i = gettimeofday(&tp,&tzp);
	gettimeofday(&tp,&tzp);
	return ( (double) tp.tv_sec + (double) tp.tv_usec * 1.e-6 );
}

extern int qcdNx;
extern int qcdNy;
extern int qcdNz;
extern int qcdNt;
extern int qcdNxy;
extern int qcdNxyz;
extern int qcdNsite;

extern int qcdMyRank;
extern int qcdNProcs;

extern int qcdNetSize[4];
extern int qcdNetPos[4];
extern int qcdRankNeighbors[8];
extern int qcdSx;
extern int qcdSy;
extern int qcdSz;
extern int qcdSt;

#define QCD_NX				8
#define QCD_NY				8
#define QCD_NZ				8
#define QCD_NT				16


#define QCD_ENORM			1.0e-16
#define QCD_NITER			200

#define QCD_CKS				0.150


void set_src(int ids,int ics,QCDComplex* Bsrc,int NtimeS)
{
	int NTsrc,NPEsrc;
	// int ist,is,ie;
	int ist;

	NTsrc = NtimeS % qcdNt;
	NPEsrc = (qcdNetSize[0] * qcdNetSize[1] * qcdNetSize[2]) * (NtimeS / qcdNt);

	QCDLA_SetConst(Bsrc,0.0,qcdNsite);

	ist = NTsrc * qcdNx * qcdNy * qcdNz;

	if(qcdMyRank == NPEsrc){
#ifdef QCD_SPINOR_3x4
		// 3x4
		Bsrc[ist].v[ids*QCD_NCOL + ics] = 1.0;
#else
		// 4x3
		// Bsrc[ist].v[ics*QCD_ND + ids] = 1.0;
		// Bsrc[ist].v[ics*QCD_ND + ids] = make_hipDoubleComplex(1.0, 0);
		Bsrc[ist + (ics*QCD_ND + ids) * qcdNsite] = make_hipDoubleComplex(1.0, 0);
#endif
	}
}

void uinit(double* pU,int lx,int ly,int lz,int lt)
{
	int i,j,x,y,z,t,is,d;
	int sx,sy,sz,st;
	int ex,ey,ez,et;
	QCDReal dt;

	sx = ((qcdNetPos[0]) * (lx)) / (qcdNetSize[0]);
	ex = ((qcdNetPos[0] + 1) * (lx)) / (qcdNetSize[0]);
	sy = ((qcdNetPos[1]) * (ly)) / (qcdNetSize[1]);
	ey = ((qcdNetPos[1] + 1) * (ly)) / (qcdNetSize[1]);
	sz = ((qcdNetPos[2]) * (lz)) / (qcdNetSize[2]);
	ez = ((qcdNetPos[2] + 1) * (lz)) / (qcdNetSize[2]);
	st = ((qcdNetPos[3]) * (lt)) / (qcdNetSize[3]);
	et = ((qcdNetPos[3] + 1) * (lt)) / (qcdNetSize[3]);

	mysrand(100);

	/* if(lx == 8 && ly == 8 && lz == 8 && lt == 16){ */
	/* 	FILE* pFile; */
	/* 	pFile = fopen("conf_08080816.txt","r"); */

	/* 	for(t=0;t<lt;t++){ */
	/* 		for(z=0;z<lz;z++){ */
	/* 			for(y=0;y<ly;y++){ */
	/* 				for(x=0;x<lx;x++){ */
	/* 					if((x >= sx && x < ex) && (y >= sy && y < ey) && (z >= sz && z < ez) && (t >= st && t < et)){ */
	/* 						for(i=0;i<4;i++){ */
	/* 							is = 18*(i*qcdNsite + (x-sx) + (y-sy)*qcdNx + (z-sz)*qcdNxy + (t-st)*qcdNxyz); */
	/* 							for(j=0;j<18;j++){ */
	/* 								fscanf(pFile,"%lf",&pU[is + j]); */
	/* 							} */
	/* 						} */
	/* 					} */
	/* 					else{ */
	/* 						for(i=0;i<4*18;i++){ */
	/* 							fscanf(pFile,"%lf",&d); */
	/* 						} */
	/* 						//fseek(pFile,4*18*8,SEEK_CUR); */
	/* 					} */
	/* 				} */
	/* 			} */
	/* 		} */
	/* 	} */

	/* 	fclose(pFile); */
	/* } */
	/* else{ */
		d = 0;
		is = 0;
		for(i=0;i<4;i++){
		    for(t=0;t<lt;t++){
			for(z=0;z<lz;z++){
 			    for(y=0;y<ly;y++){
				for(x=0;x<lx;x++){
				    if((x >= sx && x < ex) && (y >= sy && y < ey) && (z >= sz && z < ez) && (t >= st && t < et)){
					is = (x - sx) + (y - sy)*qcdNx + (z - sz)*qcdNxy + (t - st)*qcdNxyz + i*qcdNsite;
					for(j=0;j<9;j++){
					    dt = 2.0*(QCDReal)myrand()/(QCDReal)MYRAND_MAX;
					    // pU[is++] = dt - 1.0;
					    pU[is * 2] = dt - 1.0;
					    dt = 2.0*(QCDReal)myrand()/(QCDReal)MYRAND_MAX;
					    // pU[is++] = dt - 1.0;
					    pU[is * 2 + 1] = dt - 1.0;
					    is += 4*qcdNsite;
					}
				    }
				    else{
					for(j=0;j<9;j++){
					    d += myrand();
					    d += myrand();
					}
				    }
				}
			    }
			}
		    }
		}
		mysrand(d);
	/* } */
}


void cuSolve_CG(QCDComplex* dpXq,QCDComplex* dpU,QCDComplex* dpB,double CKs,double enorm,int* pNconv,double* pDiff)
{
	// static QCDSpinor* dpX = NULL;
	// static QCDSpinor* dpS = NULL;
	// static QCDSpinor* dpR = NULL;
	// static QCDSpinor* dpP = NULL;
	// static QCDSpinor* dpT = NULL;
	static QCDComplex* dpX = NULL;
	static QCDComplex* dpS = NULL;
	static QCDComplex* dpR = NULL;
	static QCDComplex* dpP = NULL;
	static QCDComplex* dpT = NULL;

	int iter,niter = 500;
	double snorm,sr,ret;
	double rr,rrp;
	double cr, bk, pap;
	int nconv = -1;

	dim3 threads(128,1,1);
	dim3 blocks(qcdNsite/threads.x,1,1);
	static double* dret = NULL;

	if(dpX == NULL){
		checkCudaErrors(hipMalloc((void**)&dret, sizeof(double)));

		checkCudaErrors(hipMalloc((void**)&dpX, sizeof(QCDComplex)*QCD_SPINOR_SIZE*qcdNsite));
		checkCudaErrors(hipMalloc((void**)&dpS, sizeof(QCDComplex)*QCD_SPINOR_SIZE*qcdNsite));
		checkCudaErrors(hipMalloc((void**)&dpR, sizeof(QCDComplex)*QCD_SPINOR_SIZE*qcdNsite));
		checkCudaErrors(hipMalloc((void**)&dpP, sizeof(QCDComplex)*QCD_SPINOR_SIZE*qcdNsite));
		checkCudaErrors(hipMalloc((void**)&dpT, sizeof(QCDComplex)*QCD_SPINOR_SIZE*qcdNsite));

		cuQCDLA_Init(qcdNsite);
	}

	cuQCDLA_Equate<<<blocks, threads>>>(dpS,dpB,qcdNsite);

	cuQCDLA_Norm(dret, (double*)dpS,qcdNsite);
	checkCudaErrors(hipMemcpy(&ret, dret, sizeof(double), hipMemcpyDeviceToHost));

	MPI_Allreduce(&ret,&sr,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);
	snorm = 1.0 / sr;

	//init
	cuQCDLA_Equate<<<blocks, threads>>>(dpR,dpS,qcdNsite);
	cuQCDLA_Equate<<<blocks, threads>>>(dpX,dpS,qcdNsite);

	cuQCDDopr_DdagD(dpS,dpU,dpX,dpT,CKs);

	cuQCDLA_MultAddScalar<<<blocks, threads>>>(dpR,dpS,-1.0,qcdNsite);

	cuQCDLA_Equate<<<blocks, threads>>>(dpP,dpR,qcdNsite);

	cuQCDLA_Norm(dret,(double*)dpR,qcdNsite);
	checkCudaErrors(hipMemcpy(&ret, dret, sizeof(double), hipMemcpyDeviceToHost));

	MPI_Allreduce(&ret,&rr,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);
	rrp = rr;

	printf("%f\n", rr);

	for(iter=0; iter < niter; iter++){
	        cuQCDDopr_DdagD(dpS,dpU,dpP,dpT,CKs);

		cuQCDLA_DotProd(dret,(double*)dpS,(double*)dpP,qcdNsite);
		checkCudaErrors(hipMemcpy(&ret, dret, sizeof(double), hipMemcpyDeviceToHost));

		MPI_Allreduce(&ret,&pap,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);
		cr = rrp/pap;

		cuQCDLA_MultAddScalar<<<blocks, threads>>>(dpX,dpP,cr,qcdNsite);
		cuQCDLA_MultAddScalar<<<blocks, threads>>>(dpR,dpS,-cr,qcdNsite);

		cuQCDLA_Norm(dret,(double*)dpR,qcdNsite);
		checkCudaErrors(hipMemcpy(&ret, dret, sizeof(double), hipMemcpyDeviceToHost));

		MPI_Allreduce(&ret,&rr,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);
		bk = rr/rrp;

		cuQCDLA_MultScalar<<<blocks, threads>>>(dpP,dpP,bk,qcdNsite);
		cuQCDLA_MultAddScalar<<<blocks, threads>>>(dpP,dpR,1.0,qcdNsite);

		rrp = rr;

		printf("%f\n", rr*snorm);

		if(rr*snorm < enorm){
			nconv = iter;
			break;
		}
	}

	if(nconv == -1 && qcdMyRank == 0){
		printf(" not converged\n");
	}

	cuQCDLA_Equate<<<blocks, threads>>>(dpXq,dpX,qcdNsite);

	cuQCDDopr_DdagD(dpR,dpU,dpX,dpT,CKs);

	cuQCDLA_MultAddScalar<<<blocks, threads>>>(dpR,dpB,-1.0,qcdNsite);

	cuQCDLA_Norm(dret,(double*)dpR,qcdNsite);
	checkCudaErrors(hipMemcpy(&ret, dret, sizeof(double), hipMemcpyDeviceToHost));
	MPI_Allreduce(&ret,&rr,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);

	*pDiff = rr;
	*pNconv = nconv;
}

int main(int argc,char** argv)
{
	int myrank=0,nprocs=1;
	int ngpus=1;
	int latsize[4],localsize[4];
	// int netSize[16],netPos[16],netDim;
	int netSize[16],netPos[16];
	int i,j,t,npIn,nsite;
	// int Niter = QCD_NITER;
	QCDComplex* pSrc;
	QCDComplex* pDest;
	QCDComplex* pGauge;

	QCDComplex* dpSrc;
	QCDComplex* dpDest;
	QCDComplex* dpGauge;

	QCDReal Enorm = QCD_ENORM;
	QCDReal Cks = QCD_CKS;
	QCDReal* pCorr;
	double tstart,tend,ttotal;
	// char* pStr;
	// int ItimeS,NtimeS,ics,ids,is,ie,ipet,it,Nconv,cnt;
	// int ics,ids,is,ipet,it,Nconv;
	int ics,ids,is,ipet,it,Nconv,iv;
	// double CorrF,Diff,rr;
	double CorrF,Diff,rr,tCorrF;
	// unsigned long flops;
	// double tt;

	latsize[0] = 0;
	latsize[1] = 0;
	latsize[2] = 0;
	latsize[3] = 0;

	// netDim = 4;
	netSize[0] = 0;
	netSize[1] = 0;
	netSize[2] = 0;
	netSize[3] = 0;

	for(i=1;i<argc;i++){
		if(argv[i][0] == 'L'){
			t = 0;
			for(j=1;j<strlen(argv[i]);j++){
				if(argv[i][j] == 'x'){
					t++;
				}
				else if(argv[i][j] >= '0' && argv[i][j] <= '9'){
					latsize[t] = 10*latsize[t] + (int)(argv[i][j] - '0');
				}
			}
		}
		else if(argv[i][0] == 'P'){
			t = 0;
			for(j=1;j<strlen(argv[i]);j++){
				if(argv[i][j] == 'x'){
					t++;
				}
				else if(argv[i][j] >= '0' && argv[i][j] <= '9'){
					netSize[t] = 10*netSize[t] + (int)(argv[i][j] - '0');
				}
			}
		}
		else if(argv[i][0] == 'G'){
		    ngpus = (int)(argv[i][1] - '0');
		}
	}

	t = 0;
	for(i=0;i<4;i++){
		if(latsize[0] == 0){
			t++;
		}
	}
	if(t > 0){
		latsize[0] = QCD_NX;
		latsize[1] = QCD_NY;
		latsize[2] = QCD_NZ;
		latsize[3] = QCD_NT;
	}

	MPI_Init(&argc,&argv);

	MPI_Comm_size(MPI_COMM_WORLD,&nprocs);
	MPI_Comm_rank(MPI_COMM_WORLD,&myrank);

	checkCudaErrors(hipSetDevice(myrank % ngpus));

	npIn = 1;
	for(i=0;i<4;i++){
		npIn *= netSize[i];

		//debug
		/* printf("netSize[%d] == %d\n", i, netSize[i]); */

	}
	if(npIn != nprocs){
		if(myrank == 0){
			printf("Number of processes is invalid\n");
		}
		return 0;
	}

	nsite = 1;
	for(i=0;i<4;i++){
		localsize[i] = latsize[i] / netSize[i];
		nsite *= localsize[i];
	}

	t = myrank;
	for(i=0;i<4;i++){
		netPos[i] = t % netSize[i];
		t /= netSize[i];
	}

	QCDDopr_Init(localsize[0],localsize[1],localsize[2],localsize[3],netSize[0],netSize[1],netSize[2],netSize[3],myrank);

	if(myrank == 0){
		printf("=============================================\n");
		printf("QCD base MPI program\n");
		printf("          Lattice size = %dx%dx%dx%d\n",latsize[0],latsize[1],latsize[2],latsize[3]);
		printf("Decomposed by %d procs : %dx%dx%dx%d\n",nprocs,netSize[0],netSize[1],netSize[2],netSize[3]);
		printf("    Local Lattice size = %dx%dx%dx%d\n",localsize[0],localsize[1],localsize[2],localsize[3]);
		printf("\n Cks = %f\n",Cks);
		printf("=============================================\n");
	}

	pGauge = (QCDComplex*)malloc(sizeof(QCDComplex) * QCD_MATRIX_SIZE * 4 * nsite + 512);
	uinit((QCDReal*)pGauge,latsize[0],latsize[1],latsize[2],latsize[3]);

	checkCudaErrors(hipMalloc((void**)&dpGauge, sizeof(QCDComplex) * QCD_MATRIX_SIZE * 4 * nsite + 512));
	checkCudaErrors(hipMemcpy(dpGauge, pGauge, sizeof(QCDComplex) * QCD_MATRIX_SIZE * 4 * nsite + 512, hipMemcpyHostToDevice));

	pSrc = (QCDComplex*)malloc(sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128);
	pDest = (QCDComplex*)malloc(sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128);

	checkCudaErrors(hipMalloc((void**)&dpSrc, sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128));
	checkCudaErrors(hipMalloc((void**)&dpDest, sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128));

	pCorr = (QCDReal*)malloc(sizeof(QCDReal) * latsize[3]);
 	for(i=0;i<latsize[3];i++){
		pCorr[i] = 0.0;
	}

	ttotal = 0.0;
	/* for(ics=0;ics<QCD_NCOL;ics++){ */
	/* 	for(ids=0;ids<QCD_ND;ids++){ */
	for(ics=0;ics<1;ics++){
		for(ids=0;ids<1;ids++){
			set_src(ids,ics,pSrc,0);

			checkCudaErrors(hipMemcpy(dpSrc, pSrc, sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128, hipMemcpyHostToDevice));
			MPI_Barrier(MPI_COMM_WORLD);
			tstart = mysecond();
			// Solve_CG(pDest,pGauge,pSrc,Cks,Enorm,&Nconv,&Diff);
			cuSolve_CG(dpDest,dpGauge,dpSrc,Cks,Enorm,&Nconv,&Diff);
			MPI_Barrier(MPI_COMM_WORLD);
			tend = mysecond() - tstart;
			ttotal += tend;

			checkCudaErrors(hipMemcpy(pDest, dpDest, sizeof(QCDComplex) * QCD_SPINOR_SIZE * nsite + 128, hipMemcpyDeviceToHost));

			if(myrank == 0){
				printf(" %3d %3d  %6d %12.4e ... %f sec\n", ics, ids, Nconv, Diff,tend);
			}

			for(i=0;i<latsize[3];i++){
				ipet = i/localsize[3];
				it = i % localsize[3];
				if(ipet == netPos[3]){
					is = it*localsize[0]*localsize[1]*localsize[2];
					// QCDLA_Norm(&CorrF,(double*)(pDest + is),localsize[0]*localsize[1]*localsize[2]);
					CorrF = 0.0;
					for (iv = 0; iv < QCD_SPINOR_SIZE; iv++) {
					    QCDLA_Norm_Simple(&tCorrF,(double*)(pDest + is + iv * qcdNsite), 2*localsize[0]*localsize[1]*localsize[2]);
					    CorrF += tCorrF;
					}
				}
				else{
					CorrF = 0.0;
				}
				MPI_Allreduce(&CorrF,&rr,1,MPI_DOUBLE_PRECISION,MPI_SUM,MPI_COMM_WORLD);
				pCorr[i] = pCorr[i] + rr;
			}
		}
	}

	if(myrank == 0){

		printf("\nPs meson correlator:\n");
		for(i=0;i<latsize[3];i++){
			printf("%d: %0.8E\n",i,pCorr[i]);
		}

		printf("\n Avg. Solver Time = %f [sec]\n",ttotal / 12);
	}

	MPI_Barrier(MPI_COMM_WORLD);

	MPI_Finalize();

	//debug
	/* printf("finish\n"); */

	return 0;
}






