#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
	Lattice QCD linear algebra routines using CUDA

	Copyright 2015 Koichi Shirahata

	Written by
		Koichi Shirahata

--------------------------------------------------------------------*/
/*--------------------------------------------------------------------
	Lattice QCD linear algebra routines

	Copyright 2009-2013 IBM Research - Tokyo, IBM Corporation

	Written by
		Jun Doi  (doichan@jp.ibm.com)

--------------------------------------------------------------------*/

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include "lib_vec.h"
#include "qcd.h"

#define QCDLA_NUM_SPIN		12

double* gtemp1;
double* gtemp2;

int Pow2(int in_size) 
{
    int pow_size = 0;
    // pow 2
    if ((in_size & (in_size - 1)) == 0) {
	pow_size = in_size;
    } else {
	int pow = 0;
	for (int s = in_size; s > 0; s >>= 1) pow++;
	pow_size = 1 << pow;
    }
    return pow_size;
}

void cuQCDLA_Init(int ns)
{
    const int n = ns*QCDLA_NUM_SPIN*2;
    int maxBlockSize = 1024;
    int blockSize = min(maxBlockSize, n);
    int gridSize = (n / blockSize) + ((n % blockSize > 0) ? 1 : 0);
    int gridSizePow = Pow2(gridSize);

    checkCudaErrors(hipMalloc((void**)&gtemp1, gridSizePow * sizeof(double)));
    checkCudaErrors(hipMalloc((void**)&gtemp2, (gridSizePow / maxBlockSize) * sizeof(double)));

    checkCudaErrors(hipMemset(gtemp1, 0, gridSizePow * sizeof(double)));
    checkCudaErrors(hipMemset(gtemp2, 0, (gridSizePow / maxBlockSize) * sizeof(double)));
}

void QCDLA_SetConst(QCDComplex* pV,QCDReal a,int ns)
{
	register hipDoubleComplex* pV0;
	int i,j;

	pV0 = pV;

	for(i=0;i<ns;i++){
	    for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = a;
		*(pV0 + j) = make_hipDoubleComplex(a, 0);
	    }
	    pV0 += QCDLA_NUM_SPIN;
	}
}




void QCDLA_Equate(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			*(pV0 + j) = *(pW0 + j);
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}

__global__ void cuQCDLA_Equate(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	// int i,j;
	int i;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV0 = pV;
	pW0 = pW;

	for(i = tid; i < ns * QCDLA_NUM_SPIN; i += stride){
	    *(pV0 + i) = *(pW0 + i);
	}
}

void QCDLA_MultScalar(QCDComplex* pV,QCDComplex* pW,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = *(pW0 + j) * PRF;
		    *(pV0 + j) = hipCmul(*(pW0 + j), make_hipDoubleComplex(PRF, 0));
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}

__global__ void cuQCDLA_MultScalar(QCDComplex* pV,QCDComplex* pW,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	// int i,j;
	int i;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV0 = pV;
	pW0 = pW;

	for(i = tid; i < ns * QCDLA_NUM_SPIN; i += stride){
	    *(pV0 + i) = hipCmul(*(pW0 + i), make_hipDoubleComplex(PRF, 0));
	}
}

void QCDLA_MultAddScalar(QCDComplex* pV,QCDComplex* pW,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) += *(pW0 + j) * PRF;
		    *(pV0 + j) = hipCadd(*(pV0 + j), hipCmul(*(pW0 + j), make_hipDoubleComplex(PRF, 0)));
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}

__global__ void cuQCDLA_MultAddScalar(QCDComplex* pV,QCDComplex* pW,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	// int i,j;
	int i;
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	pV0 = pV;
	pW0 = pW;

	for(i = tid; i < ns * QCDLA_NUM_SPIN; i += stride){
	    *(pV0 + i) = hipCadd(*(pV0 + i), hipCmul(*(pW0 + i), make_hipDoubleComplex(PRF, 0)));
	}
}


void QCDLA_Add_MultAddScalar(QCDComplex* pV,QCDComplex* pX,QCDComplex* pY,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) += (*(pX0 + j) + *(pY0 + j))*PRF;
		    *(pV0 + j) = hipCadd(*(pV0 + j), 
					hipCmul(hipCadd(*(pX0 + j), *(pY0 + j)), make_hipDoubleComplex(PRF, 0)));
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Add(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) += *(pW0 + j);
		    *(pV0 + j) = hipCadd(*(pV0 + j), *(pW0 + j));
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}

void QCDLA_Sub(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) -= *(pW0 + j);
		    *(pV0 + j) = hipCsub(*(pV0 + j), *(pW0 + j));
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Norm(double* AV,double* pV,int ns)
{
	register double* pV0;
	double r;
	// int i,j;
	int i;

	pV0 = pV;

	r = 0.0;
	for(i=0;i<ns*QCDLA_NUM_SPIN*2;i++){
		r += pV0[i] * pV0[i];
	}
	*AV = r;
}

void QCDLA_Norm_Simple(double* AV,double* pV,int ns)
{
	register double* pV0;
	double r;
	// int i,j;
	int i;

	pV0 = pV;

	r = 0.0;
	// for(i=0;i<ns*QCDLA_NUM_SPIN*2;i++){
	for(i=0;i<ns;i++){
		r += pV0[i] * pV0[i];
	}
	*AV = r;
}

void cuQCDLA_Norm(double* AV, double* pV,int ns)
{
    cuQCDLA_DotProd(AV, pV, pV, ns);
}

__global__ void cuQCDLA_Reduce_Kernel(double* pV, double* g_odata, int ns)
{
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int blockSize = blockDim.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;

    register double* pV0;
    pV0 = pV;

    while (i < ns) { sdata[tid] += pV0[i] + pV0[i + blockSize]; i += gridSize; }
    __syncthreads();
    
    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >=  512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >=  256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >=  128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >=  8) sdata[tid] += sdata[tid + 4];
	if (blockSize >=  4) sdata[tid] += sdata[tid + 2];
	if (blockSize >=  2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void cuQCDLA_DotProd_Kernel(double* pV, double* pW, double* g_odata, int ns)
{
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int blockSize = blockDim.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    sdata[tid] = 0;

    register double* pV0;
    register double* pW0;
    pV0 = pV;
    pW0 = pW;

    while (i < ns) { sdata[tid] += pV0[i] * pW0[i] + pV0[i + blockSize] * pW0[i + blockSize]; i += gridSize; }
    __syncthreads();
    
    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >=  512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >=  256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >=  128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32) {
    	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    	if (blockSize >=  8) sdata[tid] += sdata[tid + 4];
    	if (blockSize >=  4) sdata[tid] += sdata[tid + 2];
    	if (blockSize >=  2) sdata[tid] += sdata[tid + 1];
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


void QCDLA_DotProd(double* AV,double* pV,double* pW,int ns)
{
	register double* pV0;
	register double* pW0;
	double r;
	// int i,j;
	int i;

	pV0 = pV;
	pW0 = pW;

	r = 0.0;
	for(i=0;i<ns*QCDLA_NUM_SPIN*2;i++){
		r += pV0[i] * pW0[i];
	}
	*AV = r;
}

void cuQCDLA_DotProd(double* AV,double* pV,double* pW,int ns)
{
    const int n = ns*QCDLA_NUM_SPIN*2;
    const int maxBlockSize = 1024;


    int blockSize = min(maxBlockSize, n);
    int gridSize = (n / blockSize) + ((n % blockSize > 0) ? 1 : 0);
    int smemSize = blockSize * sizeof(double);

    cuQCDLA_DotProd_Kernel<<<gridSize, blockSize/2, smemSize>>>(pV, pW, gtemp1, n);
    
    int gridSizePow = Pow2(gridSize);
    if (gridSizePow > maxBlockSize) {
	blockSize = maxBlockSize;
	gridSize = gridSizePow / blockSize;
	smemSize = blockSize * sizeof(double);
	cuQCDLA_Reduce_Kernel<<<gridSize, blockSize/2, smemSize>>>(gtemp1, gtemp2, gridSizePow);

	blockSize = gridSize;
	smemSize = blockSize * sizeof(double);
	cuQCDLA_Reduce_Kernel<<<1, blockSize/2, smemSize>>>(gtemp2, AV, gridSize);
    } else if (gridSizePow > 1) {
	blockSize = gridSizePow;
	smemSize = blockSize * sizeof(double);
	cuQCDLA_Reduce_Kernel<<<1, blockSize/2, smemSize>>>(gtemp1, AV, gridSize);
    } else {
	checkCudaErrors(hipMemcpy(AV, gtemp1, sizeof(double), hipMemcpyDeviceToDevice));
    }
}


void QCDLA_MultScalar_Add(QCDComplex* pV,QCDComplex* pW,double PRF,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	int i,j;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = *(pW0 + j) + *(pV0 + j) * PRF;
		    *(pV0 + j) = hipCadd(*(pW0 + j), hipCmul(*(pV0 + j), make_hipDoubleComplex(PRF, 0)));
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_AXPBY(QCDComplex* pV,QCDComplex* pX,QCDComplex* pY,double PRF1,double PRF2,int ns)
{
        register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = PRF1 * *(pX0 + j) + PRF2 * *(pY0 + j);
		    *(pV0 + j) = hipCadd(hipCmul(make_hipDoubleComplex(PRF1, 0), *(pX0 + j)), 
					hipCmul(make_hipDoubleComplex(PRF2, 0), *(pY0 + j)));
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_AXPY(QCDComplex* pV,QCDComplex* pX,QCDComplex* pY,double a,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = a * *(pX0 + j) + *(pY0 + j);
		    *(pV0 + j) = hipCadd(hipCmul(make_hipDoubleComplex(a, 0), *(pX0 + j)), *(pY0 + j));
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
	}
}

void QCDLA_AXMY(QCDComplex* pV,QCDComplex* pX,QCDComplex* pY,double a,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = a * *(pX0 + j) - *(pY0 + j);
		    *(pV0 + j) = hipCsub(hipCmul(make_hipDoubleComplex(a, 0), *(pX0 + j)), *(pY0 + j));
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
	}
}



void QCDLA_AXPBYPZ(QCDComplex* pV,QCDComplex* pX,QCDComplex* pY,QCDComplex* pZ,double a,double b,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	register hipDoubleComplex* pZ0;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;
	pZ0 = pZ;

	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = a * *(pX0 + j) + b * *(pY0 + j) + *(pZ0 + j);
		    *(pV0 + j) = hipCadd(hipCadd(hipCmul(make_hipDoubleComplex(a, 0), *(pX0 + j)), 
					       hipCmul(make_hipDoubleComplex(b, 0), *(pY0 + j))), 
					*(pZ0 + j));
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
		pZ0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_AXPY_Norm(QCDComplex* pV,double* AV,QCDComplex* pX,QCDComplex* pY,double a,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pX0;
	register hipDoubleComplex* pY0;
	double* pN;
	double r;
	int i,j;

	pV0 = pV;
	pX0 = pX;
	pY0 = pY;

	r = 0.0;
	for(i=0;i<ns;i++){
		for(j=0;j<QCDLA_NUM_SPIN;j++){
			// *(pV0 + j) = a * *(pX0 + j) + *(pY0 + j);
		    *(pV0 + j) = hipCadd(hipCmul(make_hipDoubleComplex(a, 0), *(pX0 + j)), *(pY0 + j));
			pN = (double*)(pV0 + j);
			r += pN[0] * pN[0] + pN[1] * pN[1];
		}
		pV0 += QCDLA_NUM_SPIN;
		pX0 += QCDLA_NUM_SPIN;
		pY0 += QCDLA_NUM_SPIN;
	}
	*AV = r;
}


#ifdef QCD_SPINOR_3x4
/*

0: w(1,1) w(2,1) 

1: w(3,1) w(1,2) 

2: w(2,2) w(3,2)


3: w(1,3) w(2,3) 

4: w(3,3) w(1,4) 

5: w(2,4) w(3,4)

*/


void QCDLA_MultGamma5(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register double _Complex* pV0;
	register double _Complex* pW0;
	register double _Complex v0;
	register double _Complex v1;
	register double _Complex v2;
	register double _Complex v3;
	int i,j,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			v0 = *(pW0     + k);
			v1 = *(pW0 + 3 + k);
			v2 = *(pW0 + 6 + k);
			v3 = *(pW0 + 9 + k);
			*(pV0     + k) = v2;
			*(pV0 + 3 + k) = v3;
			*(pV0 + 6 + k) = v0;
			*(pV0 + 9 + k) = v1;
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Proj_P(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register double _Complex* pV0;
	register double _Complex* pW0;
	double p0,p1;
	int i,j,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			p0 = 0.5*(*(pW0 + k    ) + *(pW0 + k + 6));
			p1 = 0.5*(*(pW0 + k + 3) + *(pW0 + k + 9));
			*(pV0 + k    ) = p0;
			*(pV0 + k + 3) = p1;
			*(pV0 + k + 6) = p0;
			*(pV0 + k + 9) = p1;
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Proj_M(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register double _Complex* pV0;
	register double _Complex* pW0;
	double p0,p1;
	int i,j,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			p0 = 0.5*(*(pW0 + k    ) - *(pW0 + k + 6));
			p1 = 0.5*(*(pW0 + k + 3) - *(pW0 + k + 9));
			*(pV0 + k    ) = p0;
			*(pV0 + k + 3) = p1;
			*(pV0 + k + 6) = -p0;
			*(pV0 + k + 9) = -p1;
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}




#else	//QCD_SPINOR_3x4
void QCDLA_MultGamma5(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	register hipDoubleComplex v0;
	register hipDoubleComplex v1;
	// int i,j,k;
	int i,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			v0 = *(pW0 + k*4    );
			v1 = *(pW0 + k*4 + 1);
			*(pV0 + k*4    ) = *(pW0 + k*4 + 2);
			*(pV0 + k*4 + 1) = *(pW0 + k*4 + 3);
			*(pV0 + k*4 + 2) = v0;
			*(pV0 + k*4 + 3) = v1;
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Proj_P(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	hipDoubleComplex p0,p1;
	// int i,j,k;
	int i,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			// p0 = 0.5*(*(pW0 + k*4    ) + *(pW0 + k*4 + 2));
			// p1 = 0.5*(*(pW0 + k*4 + 1) + *(pW0 + k*4 + 3));
		    p0 = hipCmul(make_hipDoubleComplex(0.5, 0), 
				hipCadd(*(pW0 + k*4    ), *(pW0 + k*4 + 2)));
		    p1 = hipCmul(make_hipDoubleComplex(0.5, 0), 
				hipCadd(*(pW0 + k*4 + 1), *(pW0 + k*4 + 3)));
			*(pV0 + k*4    ) = p0;
			*(pV0 + k*4 + 1) = p1;
			*(pV0 + k*4 + 2) = p0;
			*(pV0 + k*4 + 3) = p1;
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}


void QCDLA_Proj_M(QCDComplex* pV,QCDComplex* pW,int ns)
{
	register hipDoubleComplex* pV0;
	register hipDoubleComplex* pW0;
	hipDoubleComplex p0,p1;
	// int i,j,k;
	int i,k;

	pV0 = pV;
	pW0 = pW;

	for(i=0;i<ns;i++){
		for(k=0;k<3;k++){
			// p0 = 0.5*(*(pW0 + k*4    ) - *(pW0 + k*4 + 2));
			// p1 = 0.5*(*(pW0 + k*4 + 1) - *(pW0 + k*4 + 3));
		    p0 = hipCmul(make_hipDoubleComplex(0.5, 0), 
				hipCsub(*(pW0 + k*4    ), *(pW0 + k*4 + 2)));
		    p1 = hipCmul(make_hipDoubleComplex(0.5, 0),
				hipCsub(*(pW0 + k*4 + 1), *(pW0 + k*4 + 3)));
			*(pV0 + k*4    ) = p0;
			*(pV0 + k*4 + 1) = p1;
			// *(pV0 + k*4 + 2) = -p0;
			// *(pV0 + k*4 + 3) = -p1;
			*(pV0 + k*4 + 2) = hipCsub(*(pV0 + k*4 + 2), p0);
			*(pV0 + k*4 + 3) = hipCsub(*(pV0 + k*4 + 3), p1);
		}
		pV0 += QCDLA_NUM_SPIN;
		pW0 += QCDLA_NUM_SPIN;
	}
}

#endif


